#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <time.h>

#define THREAD 1024

using namespace std;

__global__
void add_kernel(long * d_a, long * d_tmp, long k, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	i=(i+1)*(n / (gridDim.x*blockDim.x))-1;
	k=k*(n / (gridDim.x*blockDim.x));
	if (i + k < n) {
		d_tmp[i + k] = d_a[i + k] + d_a[i];
	}
}

__global__
void local_sum(long * d_a, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = i*(n / (gridDim.x*blockDim.x));
	int k = (i+1)*(n / (gridDim.x*blockDim.x));
	for (;j < k-1;j++) {
		d_a[j + 1] = d_a[j + 1] + d_a[j];
	}
}

__global__
void local_add(long * d_a, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i == 0) return;
	int j = i*(n / (gridDim.x*blockDim.x))-1;
	for (int k = 1; k < (n / (gridDim.x*blockDim.x)); k++) {
		d_a[j + k] = d_a[j] + d_a[j + k];
	}
}

int main() {
	
	for (int k = 11; k <= 26; k++)
	{
		long *a, *d_a, *d_tmp;
		long n = 1 << k;
		int thread = 1024, block = 2;
		//Allocate memory on CPU
		a = (long *)malloc(n * sizeof(long));

		//Initialize values
		for (long i = 0; i < n; i++) {
			a[i] =  1;
		}

		//Allocate memory on GPU
		hipMalloc(&d_a, n * sizeof(long));
		hipMalloc(&d_tmp, n * sizeof(long)); //To hold temporary results

		//Copy content from CPU to GPU
		hipMemcpy(d_a, a, n * sizeof(long), hipMemcpyHostToDevice);

		//Copy content in to temporary array
		hipMemcpy(d_tmp, d_a, n * sizeof(long), hipMemcpyDeviceToDevice);

		clock_t begin = clock();
		//First pass
		local_sum << <block, thread >> > (d_a, n);
		hipMemcpy(d_tmp, d_a, n * sizeof(long), hipMemcpyDeviceToDevice);
		for (long p = 0; p <=log2l(2*thread)-1; p++) {
			add_kernel << <block, thread >> > (d_a, d_tmp, 1 << p, n);
			hipMemcpy(d_a, d_tmp, n * sizeof(long), hipMemcpyDeviceToDevice);
		}
		local_add << <block, thread >> > (d_a, n);

		//Second Pass
		local_sum << <block, thread >> > (d_a, n);
		hipMemcpy(d_tmp, d_a, n * sizeof(long), hipMemcpyDeviceToDevice);
		for (long p = 0; p <= log2l(2 * thread) - 1; p++) {
			add_kernel << <block, thread >> > (d_a, d_tmp, 1 << p, n);
			hipMemcpy(d_a, d_tmp, n * sizeof(long), hipMemcpyDeviceToDevice);
		}
		local_add << <block, thread >> > (d_a, n);

		hipDeviceSynchronize();

		clock_t end = clock();
		double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
		cout<<"The running time is " << time_spent << " milliseconds."<<endl;
		//Copy results back to CPU
		hipMemcpy(a, d_a, n * sizeof(long), hipMemcpyDeviceToHost);
	
		//Verify results
		long *b;
		b = (long *)malloc(n * sizeof(long));
		//Verify results
		for (int i = 0; i < n; i++)
		{
		    b[i] = 1;
		}
		for (int i = 1; i < n; i++)
		{
		    b[i] = b[i] + b[i - 1];
		}
		for (int i = 1; i < n; i++)
		{
		    b[i] = b[i] + b[i - 1];
		}
		for (int i = 1; i < n; i++)
		{
		    if (a[i] != b[i])
		    {
			cout << "Incorrect Result " << i << " " << a[i] <<" "<< b[i] << endl;
			break;
		    }
		}

		//Free memory on CPU
		free(a);

		//Free memory on GPU
		hipFree(d_a);
		hipFree(d_tmp);
	}
	return 0;
}
