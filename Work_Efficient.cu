#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""	
#include <iostream>
#include <cmath>
#include <ctime>
#include <time.h>
#include <stdio.h>

#define POWER 28
#define THREAD 1024

using namespace std;

__global__
void upsweep_add(double * d_a, double * d_tmp, long k, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	long index = i * k - 1;
	if (index >= 0 && index < n) {
		d_tmp[index] = d_a[index] + d_a[index-(k/2)];
	}
}

__global__
void downsweep_add(double * d_a, double * d_tmp, long k, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	long index = i * k - 1;
	if (index >= 0 && index < n) {
		double t = d_a[index-(k/2)];
		d_tmp[index-(k/2)] = d_a[index];
		d_tmp[index] = d_a[index] + t;
	}
}

__global__
void vector_add(double * d_a, double * d_o, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
		d_a[i] = d_o[i] + d_a[i];
}

__host__
void prefix_sum(double * d_a, double * d_tmp, double * d_o, long n) {

	//Copy content in to temporary array
	hipMemcpy(d_tmp, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	//Launch kernel log n times 
	for (long p = 1; p <= POWER; p++) {
		upsweep_add << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_tmp, 1 << p, n);
		hipMemcpy(d_a, d_tmp, n * sizeof(double), hipMemcpyDeviceToDevice);
	}
	hipMemset(d_a + (n - 1), 0, sizeof(double));
	hipMemcpy(d_tmp, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	for (long p = 0; p < POWER; p++) {
		downsweep_add << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_tmp, 1 << (POWER - p), n);
		hipMemcpy(d_a, d_tmp, n * sizeof(double), hipMemcpyDeviceToDevice);
	}

	vector_add << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_o, n);
}

__host__
double verify_answers(double *a, double * b, long n) {
	double  *v;

	v = (double *)malloc(n * sizeof(double));

	for (int i = 0; i < n; i++) {
		v[i] = a[i];
	}

	for (int i = 1; i < n; i++) {
		v[i] = v[i] + v[i - 1];
	}

	for (int i = 1; i < n; i++) {
		v[i] = v[i] + v[i - 1];
	}
	double maxError = 0;
	for (int i = 0; i < n; i++) {
		maxError = fmax(maxError, fabs(v[i] - b[i]));
	}
	return maxError / v[n - 1];
}

__host__
void compute_answers(double * a, double * b, long n) {
	double *d_a, *d_tmp, *d_o;

	//Allocate memory on GPU
	hipMalloc(&d_a, n * sizeof(double));
	hipMalloc(&d_tmp, n * sizeof(double)); //To hold temporary results
	hipMalloc(&d_o, n * sizeof(double));

	//Copy content from CPU to GPU
	hipMemcpy(d_a, a, n * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(d_o, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	//First Prefix Sum
	prefix_sum(d_a, d_tmp, d_o, n);

	//Copy content in to temporary array
	hipMemcpy(d_tmp, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);
	hipMemcpy(d_o, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	//Second Prefix Sum
	prefix_sum(d_a, d_tmp, d_o, n);

	//Copy results back to CPU
	hipMemcpy(b, d_a, n * sizeof(double), hipMemcpyDeviceToHost);

	//Free memory on GPU
	hipFree(d_a);
	hipFree(d_tmp);
	hipFree(d_o);
}

timespec time_diff(timespec start, timespec end)
{
    timespec temp;
    if ((end.tv_nsec-start.tv_nsec)<0) {
        temp.tv_sec = end.tv_sec-start.tv_sec-1;
        temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
    } else {
        temp.tv_sec = end.tv_sec-start.tv_sec;
        temp.tv_nsec = end.tv_nsec-start.tv_nsec;
    }
    return temp;
}

int main() {
    printf("power seconds nanoseconds error\n");
    for (int power = 1; power < POWER; power++){
        double *a, *b;
        long n = 1 << power;

        //Allocate memory on CPU
        a = (double *)malloc(n * sizeof(double));
        b = (double *)malloc(n * sizeof(double));
        srand(clock());
        //Initialize values
        for (long i = 0; i < n; i++) {
            a[i] = ((double)(rand() % n)) / 100;
        }
        
        struct timespec start, end, difference;
        clock_gettime(CLOCK_MONOTONIC, &start);
        //Compute Answers
        compute_answers(a, b, n);
        clock_gettime(CLOCK_MONOTONIC, &end);
        difference = time_diff(start,end);
        printf("%d %d %ld %e\n",power+1, difference.tv_sec, difference.tv_nsec, verify_answers(a,b,n));

        //Free memory on CPU
        free(a);
        free(b);
    }

	return 0;
}
