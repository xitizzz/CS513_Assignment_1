#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <time.h> //for clock_gettime, high precision timer
#include <stdio.h> //for printf

#define POWER 28
#define THREAD 1024

using namespace std;

__global__
void add_kernel(double * d_a, double * d_tmp, long k, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i + k < n) {
		d_tmp[i + k] = d_a[i + k] + d_a[i];
	}
}

__host__
void compute_answers(double * a, double * b, long n) {
	double  *d_a, *d_tmp;

	//Allocate memory on GPU
	hipMalloc(&d_a, n * sizeof(double));
	hipMalloc(&d_tmp, n * sizeof(double)); //To hold temporary results

										  //Copy content from CPU to GPU
	hipMemcpy(d_a, a, n * sizeof(double), hipMemcpyHostToDevice);

	//Copy content in to temporary array
	hipMemcpy(d_tmp, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	//First pass
	//Launch kernel log n times 
	for (long p = 0; p <= POWER; p++) {
		add_kernel << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_tmp, 1 << p, n);
		hipMemcpy(d_a, d_tmp, n * sizeof(double), hipMemcpyDeviceToDevice);
	}

	hipMemcpy(d_tmp, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	//Second pass
	//Launch kernel log n times 
	for (long p = 0; p <= POWER; p++) {
		add_kernel << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_tmp, 1 << p, n);
		hipMemcpy(d_a, d_tmp, n * sizeof(double), hipMemcpyDeviceToDevice);
	}

	//Copy results back to CPU
	hipMemcpy(b, d_a, n * sizeof(double), hipMemcpyDeviceToHost);

	//Free memory on GPU
	hipFree(d_a);
	hipFree(d_tmp);
}

__host__
double verify_answers(double *a, double * b, long n) {
	double  *v;

	v = (double *)malloc(n * sizeof(double));

	for (int i = 0; i < n; i++) {
		v[i] = a[i];
	}

	for (int i = 1; i < n; i++) {
		v[i] = v[i] + v[i - 1];
	}

	for (int i = 1; i < n; i++) {
		v[i] = v[i] + v[i - 1];
	}
	double maxError = 0;
	for (int i = 0; i < n; i++) {
		maxError = fmax(maxError, fabs(v[i] - b[i]));
	}
	return maxError/v[n-1];
}

timespec time_diff(timespec start, timespec end)
{
    timespec temp;
    if ((end.tv_nsec-start.tv_nsec)<0) {
        temp.tv_sec = end.tv_sec-start.tv_sec-1;
        temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
    } else {
        temp.tv_sec = end.tv_sec-start.tv_sec;
        temp.tv_nsec = end.tv_nsec-start.tv_nsec;
    }
    return temp;
}

int main() {
    printf("power seconds nanoseconds error\n");
    for (int power = 1; power < POWER; power++){
        double *a, *b;
        long n = 1 << power;

        //Allocate memory on CPU
        a = (double *)malloc(n * sizeof(double));
        b = (double *)malloc(n * sizeof(double));

        //Initialize values
        for (long i = 0; i < n; i++) {
            a[i] = ((double)(rand() % n)) / 100;
        }


        //Compute Answers
        compute_answers(a, b, n);

        double error_amt = 0;
        struct timespec start, end, difference;
        clock_gettime(CLOCK_MONOTONIC, &start);
        error_amt = verify_answers(a,b,n);
        clock_gettime(CLOCK_MONOTONIC, &end);

        difference = time_diff(start,end);

        printf("%d %d %ld %e\n",power+1, difference.tv_sec, difference.tv_nsec, error_amt);

        //Free memory on CPU
        free(a);
        free(b);
    }
	return 0;
}
