#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>

#define POWER 24
#define THREAD 1024

using namespace std;

__global__
void add_kernel(double * d_a, double * d_tmp, long k, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i + k < n) {
		d_tmp[i + k] = d_a[i + k] + d_a[i];
	}
}

__host__
void compute_answers(double * a, double * b, long n) {
	double  *d_a, *d_tmp;

	//Allocate memory on GPU
	hipMalloc(&d_a, n * sizeof(double));
	hipMalloc(&d_tmp, n * sizeof(double)); //To hold temporary results

										  //Copy content from CPU to GPU
	hipMemcpy(d_a, a, n * sizeof(double), hipMemcpyHostToDevice);

	//Copy content in to temporary array
	hipMemcpy(d_tmp, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	//First pass
	//Launch kernel log n times 
	for (long p = 0; p <= POWER; p++) {
		add_kernel << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_tmp, 1 << p, n);
		hipMemcpy(d_a, d_tmp, n * sizeof(double), hipMemcpyDeviceToDevice);
	}

	hipMemcpy(d_tmp, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	//Second pass
	//Launch kernel log n times 
	for (long p = 0; p <= POWER; p++) {
		add_kernel << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_tmp, 1 << p, n);
		hipMemcpy(d_a, d_tmp, n * sizeof(double), hipMemcpyDeviceToDevice);
	}

	//Copy results back to CPU
	hipMemcpy(b, d_a, n * sizeof(double), hipMemcpyDeviceToHost);

	//Free memory on GPU
	hipFree(d_a);
	hipFree(d_tmp);
}

__host__
double verify_answers(double *a, double * b, long n) {
	double  *v;

	v = (double *)malloc(n * sizeof(double));

	for (int i = 0; i < n; i++) {
		v[i] = a[i];
	}

	for (int i = 1; i < n; i++) {
		v[i] = v[i] + v[i - 1];
	}

	for (int i = 1; i < n; i++) {
		v[i] = v[i] + v[i - 1];
	}
	double maxError = 0;
	for (int i = 0; i < n; i++) {
		maxError = fmax(maxError, fabs(v[i] - b[i]));
	}
	return maxError/v[n-1];
}

int main() {

	double *a, *b;
	long n = 1 << POWER;

	//Allocate memory on CPU
	a = (double *)malloc(n * sizeof(double));
	b = (double *)malloc(n * sizeof(double));

	//Initialize values
	for (long i = 0; i < n; i++) {
		a[i] = ((double)(rand() % n)) / 100;
	}

	//Compute Answers
	compute_answers(a, b, n);

	//Verify Answers
	cout<<"Error margin: " <<verify_answers(a, b, n)<<endl;

	//Free memory on CPU
	free(a);
	free(b);

	return 0;
}
