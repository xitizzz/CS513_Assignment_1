#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>

#define POWER 25
#define THREAD 1024

using namespace std;

__global__
void add_kernel(long * d_a, long * d_tmp, long k, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i + k < n) {
		d_tmp[i + k] = d_a[i + k] + d_a[i];
	}
}

int main() {
	long *a, *d_a, *d_tmp;
	long n = 1 << POWER;

	//Allocate memory on CPU
	a = (long *)malloc(n * sizeof(long));

	//Initialize values
	for (long i = 0; i < n; i++) {
		a[i] =i+1;
	}

	//Allocate memory on GPU
	hipMalloc(&d_a, n * sizeof(long));
	hipMalloc(&d_tmp, n * sizeof(long)); //To hold temporary results

	//Copy content from CPU to GPU
	hipMemcpy(d_a, a, n * sizeof(long), hipMemcpyHostToDevice);

	//Copy content in to temporary array
	hipMemcpy(d_tmp, d_a, n * sizeof(long), hipMemcpyDeviceToDevice);

	//First pass
	//Launch kernel log n times 
	for (long p = 0; p <= POWER; p++) {
		add_kernel << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_tmp, 1 << p, n);
		hipMemcpy(d_a, d_tmp, n * sizeof(long), hipMemcpyDeviceToDevice);
	}

	hipMemcpy(d_tmp, d_a, n * sizeof(long), hipMemcpyDeviceToDevice);
	
	//Second pass
	//Launch kernel log n times 
	for (long p = 0; p <= POWER; p++) {
		add_kernel << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_tmp, 1 << p, n);
		hipMemcpy(d_a, d_tmp, n * sizeof(long), hipMemcpyDeviceToDevice);
	}

	//Copy results back to CPU
	hipMemcpy(a, d_a, n * sizeof(long), hipMemcpyDeviceToHost);

	//Verify results
	for (long i = 0; i < n; i++) {
		//cout << a[i] << "\t";
		if (a[i] != 2 * (i + 1)) {
			//cout << "Incorrect Result " << i << a[i] << endl;
			//break;
		}
	}

	//Free memory on CPU
	free(a);

	//Free memory on GPU
	hipFree(d_a);
	hipFree(d_tmp);

	return 0;
}
