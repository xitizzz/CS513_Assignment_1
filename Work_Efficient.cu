#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""	
#include <iostream>
#include <cmath>
#include <ctime>

#define POWER 24
#define THREAD 1024

using namespace std;

__global__
void upsweep_add(double * d_a, double * d_tmp, long k, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	long index = i * k - 1;
	if (index >= 0 && index < n) {
		d_tmp[index] = d_a[index] + d_a[index-(k/2)];
	}
}

__global__
void downsweep_add(double * d_a, double * d_tmp, long k, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	long index = i * k - 1;
	if (index >= 0 && index < n) {
		double t = d_a[index-(k/2)];
		d_tmp[index-(k/2)] = d_a[index];
		d_tmp[index] = d_a[index] + t;
	}
}

__global__
void vector_add(double * d_a, double * d_o, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
		d_a[i] = d_o[i] + d_a[i];
}

__host__
void prefix_sum(double * d_a, double * d_tmp, double * d_o, long n) {

	//Copy content in to temporary array
	hipMemcpy(d_tmp, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	//Launch kernel log n times 
	for (long p = 1; p <= POWER; p++) {
		upsweep_add << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_tmp, 1 << p, n);
		hipMemcpy(d_a, d_tmp, n * sizeof(double), hipMemcpyDeviceToDevice);
	}
	hipMemset(d_a + (n - 1), 0, sizeof(double));
	hipMemcpy(d_tmp, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	for (long p = 0; p < POWER; p++) {
		downsweep_add << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_tmp, 1 << (POWER - p), n);
		hipMemcpy(d_a, d_tmp, n * sizeof(double), hipMemcpyDeviceToDevice);
	}

	vector_add << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_o, n);
}

__host__
double verify_answers(double *a, double * b, long n) {
	double  *v;

	v = (double *)malloc(n * sizeof(double));

	for (int i = 0; i < n; i++) {
		v[i] = a[i];
	}

	for (int i = 1; i < n; i++) {
		v[i] = v[i] + v[i - 1];
	}

	for (int i = 1; i < n; i++) {
		v[i] = v[i] + v[i - 1];
	}
	double maxError = 0;
	for (int i = 0; i < n; i++) {
		maxError = fmax(maxError, fabs(v[i] - b[i]));
	}
	return maxError / v[n - 1];
}

__host__
void compute_answers(double * a, double * b, long n) {
	double *d_a, *d_tmp, *d_o;

	//Allocate memory on GPU
	hipMalloc(&d_a, n * sizeof(double));
	hipMalloc(&d_tmp, n * sizeof(double)); //To hold temporary results
	hipMalloc(&d_o, n * sizeof(double));

	//Copy content from CPU to GPU
	hipMemcpy(d_a, a, n * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(d_o, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	//First Prefix Sum
	prefix_sum(d_a, d_tmp, d_o, n);

	//Copy content in to temporary array
	hipMemcpy(d_tmp, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);
	hipMemcpy(d_o, d_a, n * sizeof(double), hipMemcpyDeviceToDevice);

	//Second Prefix Sum
	prefix_sum(d_a, d_tmp, d_o, n);

	//Copy results back to CPU
	hipMemcpy(b, d_a, n * sizeof(double), hipMemcpyDeviceToHost);

	//Free memory on GPU
	hipFree(d_a);
	hipFree(d_tmp);
	hipFree(d_o);
}

int main() {
	double *a, *b;
	long n = 1 << POWER;

	//Allocate memory on CPU
	a = (double *)malloc(n * sizeof(double));
	b = (double *)malloc(n * sizeof(double));
	srand(clock());
	//Initialize values
	for (long i = 0; i < n; i++) {
		a[i] = ((double)(rand() % n)) / 100;
	}
	
	clock_t begin = clock();
	//Compute Answers
	compute_answers(a, b, n);
	clock_t end = clock();

	cout << "Time:" << ((double)(end - begin) / CLOCKS_PER_SEC) * 1000 << endl;

	//Verify Answers
	cout << "Error margin: " << verify_answers(a, b, n) << endl;

	//Free memory on CPU
	free(a);
	free(b);


	return 0;
}
