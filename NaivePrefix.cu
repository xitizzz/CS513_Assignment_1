#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <time.h>

#define POWER 25
#define THREAD 1024

using namespace std;

__global__
void add_kernel(long * d_a, long * d_tmp, long k, long n) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i + k < n) {
		d_tmp[i + k] = d_a[i + k] + d_a[i];
	}
}

int main() {
	long *a, *d_a, *d_tmp;
	long n = 1 << POWER;

	//Allocate memory on CPU
	a = (long *)malloc(n * sizeof(long));

	//Initialize values
	for (long i = 0; i < n; i++) {
		a[i] = i+1;
	}

	//Allocate memory on GPU
	hipMalloc(&d_a, n * sizeof(long));
	hipMalloc(&d_tmp, n * sizeof(long)); //To hold temporary results

	//Copy content from CPU to GPU
	hipMemcpy(d_a, a, n * sizeof(long), hipMemcpyHostToDevice);

	//Copy content in to temporary array
	hipMemcpy(d_tmp, d_a, n * sizeof(long), hipMemcpyDeviceToDevice);

	clock_t begin = clock();
	//First pass
	//Launch kernel log n times 
	for (long p = 0; p <= POWER; p++) {
		add_kernel <<<(n + THREAD - 1) / THREAD, THREAD >>> (d_a, d_tmp, 1 << p, n);
		hipMemcpy(d_a, d_tmp, n * sizeof(long), hipMemcpyDeviceToDevice);
	}

	hipMemcpy(d_tmp, d_a, n * sizeof(long), hipMemcpyDeviceToDevice);
	
	//Second pass
	//Launch kernel log n times 
	for (long p = 0; p <= POWER; p++) {
		add_kernel << <(n + THREAD - 1) / THREAD, THREAD >> > (d_a, d_tmp, 1 << p, n);
		hipMemcpy(d_a, d_tmp, n * sizeof(long), hipMemcpyDeviceToDevice);
	}
	
	hipDeviceSynchronize();

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
	cout<<"The running time is " << time_spent << " milliseconds."<<endl;

	//Copy results back to CPU
	hipMemcpy(a, d_a, n * sizeof(long), hipMemcpyDeviceToHost);
	long *b;
	b = (long *)malloc(n * sizeof(long));
	//Verify results
	for (int i = 0; i < n; i++)
	{
	    b[i] = i + 1;
	}
	for (int i = 1; i < n; i++)
	{
	    b[i] = b[i] + b[i - 1];
	}
	for (int i = 1; i < n; i++)
	{
	    b[i] = b[i] + b[i - 1];
	}
	for (int i = 1; i < n; i++)
	{
	    if (a[i] != b[i])
	    {
		cout << "Incorrect Result " << a[i] <<" "<< b[i] << endl;
		break;
            }
	}
	//Free memory on CPU
	free(a);

	//Free memory on GPU
	hipFree(d_a);
	hipFree(d_tmp);

	return 0;
}
